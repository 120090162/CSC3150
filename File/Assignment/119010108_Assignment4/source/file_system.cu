﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 mod_time = 0;            // store the modify time 
__device__ __managed__ u32 crea_time = 0;           // store the create time
__device__ __managed__ u32 block_pos = 0;           // store the next block location 
__device__ __managed__ u32 FCB_pos = 4096;
__device__ __managed__ u32 current_FCB_pos = 4096; // store the next FCB entry location 

// node structure in fcb
// [0,19] --> filename 
// [20,21] --> file create time 
// [22,23] --> file modify time 
// [24,27] --> file start block
// [28,31] --> file size  


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

}


__device__ u32 IsFileExist(FileSystem * fs, char *s){
  /* if exist return the FCB position */
  int flag;

  for (int i = 4096; i < 4096 + 32*1024 - 1; i = i + 32){  
    flag = 0;

    /* if size is 0, then stop traverse */
    if (fs->volume[i+28] == 0 &&  fs->volume[i+29] == 0 && fs->volume[i+30] == 0 && fs->volume[i+31] ==0 ){
      break;
    }

    /* search the filename */
    for (int j = 0; j < 20; ++j){
      if (fs->volume[j+i] != s[j]){
        flag =1;
        break;
      }
    }


    /* if exist, return the FCB block location  */
    if (flag == 0) return i;

  }

  /* not exist */
  return -1;
}

__device__ bool IsEnoughSpace(FileSystem * fs, u32 fp, u32 size){

  u32 final_block = fp + (size - 1) / 32 ;

  /* the final block position in super */
  u32 super_block_pos = final_block / 8;
  u32 super_block_remain = final_block % 8; 
  u32 temp = fs->volume[super_block_pos] >> super_block_remain;
  return temp % 2 == 0;
}

__device__ u32 Update(FileSystem * fs, u32 fp, u32 size ){

  u32 pos = fs->FILE_BASE_ADDRESS + fp * 32; // the intial position for file 
  u32 required_size = ((size -1)/32 + 1)*32;   // the required space for file including internal fragmentation 

  /* if write the file occupy other file's space, move them */
  while ( (fs->volume[pos + required_size] != 0 || (pos+required_size)%32 != 0) && pos + size < fs->STORAGE_SIZE  ){
    fs->volume[pos] = fs->volume[pos + required_size];
    fs->volume[pos + required_size] = 0;
    pos++;
  }

  /* update the block */
  for (int i = 0; i < block_pos/8 + 1; i++){
    // set it all to zero
    fs->volume[i] = 0;
  }
  block_pos = block_pos - (size-1)/32 -1;
  u32 whole_block = block_pos/8;
  u32 remain = block_pos%8;

  // set the block before to 511(111111111)
  for (int i = 0; i < whole_block && i < fs->SUPERBLOCK_SIZE ; i++) {
		fs->volume[i] = 511;
	}

  // set the remain bit to 0
  for (int i = 0; i < remain; i++) {
		fs->volume[whole_block] = fs->volume[whole_block] + (1 << i); // modify one bit 
	}

  /* modifty the fcb */
  u32 fcb_temp_pos;

  for (int i = 4096; i < 36863; i = i + 32){
    if (fs->volume[i+28] == 0 && fs->volume[i+29] && fs->volume[i+30] ==  0 && fs->volume[i=31] == 0){
      break; // search till empty 
    }
    fcb_temp_pos = (fs->volume[i+24] << 24) + (fs->volume[i+25] << 16)
                  + (fs->volume[i+26] << 8) +  (fs->volume[i+27]);
    if (fcb_temp_pos > fp){
      // clear the external space 
      fcb_temp_pos = fcb_temp_pos - (size-1)/32 - 1;
      fs->volume[i + 24] = fcb_temp_pos >> 24;
      fs->volume[i + 25] = fcb_temp_pos >> 16;
      fs->volume[i + 26] = fcb_temp_pos >> 8;
      fs->volume[i + 27] = fcb_temp_pos;
    }
  }
}


__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
  u32 ISExist = IsFileExist(fs, s);
  
  /* file not exist */
  if (ISExist == -1){

    /* read mode */
    if (op == 0){
      printf("Open Error: You can not Read file that doesn't exist! \n");
      return -1;
    }

    /* write mode */
    if (op == 1){

      current_FCB_pos = FCB_pos;

      /* store the file name */
      for (int i = 0; i < 20; i++){
        fs->volume[current_FCB_pos + i] = s[i]; 
      }

      /* store the create time */
      fs->volume[FCB_pos + 20] = crea_time >> 8;
      fs->volume[FCB_pos + 21] = crea_time;

      /* store the modify time */
      fs->volume[FCB_pos + 22] = mod_time >> 8;
      fs->volume[FCB_pos + 23] = mod_time;

      /* store the start block */
      fs->volume[FCB_pos + 24] = block_pos >> 24;
      fs->volume[FCB_pos + 25] = block_pos >> 16;
      fs->volume[FCB_pos + 26] = block_pos >> 8;
      fs->volume[FCB_pos + 27] = block_pos;

      /* update the date */
      crea_time++;
      mod_time++;

      /* update the fcb */
      FCB_pos+=32;

      return block_pos;
    }
  }

  /* file exist */
  else{
    
    /* assign the position to current */
    current_FCB_pos = IsFileExist(fs,s);
    u32 start_block = (fs->volume[current_FCB_pos+24] << 24) + (fs->volume[current_FCB_pos+25] << 16) 
                  +(fs->volume[current_FCB_pos+26] << 8) + (fs->volume[current_FCB_pos+27] );

    /* write mode */
    if (op == 1){
      u32 filesize = (fs->volume[current_FCB_pos+28] << 24) + (fs->volume[current_FCB_pos+29] << 16)
                    +(fs->volume[current_FCB_pos+30] << 8) + (fs->volume[current_FCB_pos+31]);

      /* clean the old file in disk */
      for (int i = 0; i < filesize; ++i){
        fs->volume[fs->FILE_BASE_ADDRESS + start_block * 32 + i] = 0;
      }
      
      /* update the super block */
      for (int i = 0; i < (filesize -1)/32 + 1; i++){
        u32 super_block = start_block + i;
        int shiftnum = super_block % 8;
        fs->volume[super_block/8] = fs->volume[super_block/8] - (1 << shiftnum); // modify one bit
      }

      
      /* update fcb */
      fs->volume[current_FCB_pos + 22] =  mod_time >> 8;
      fs->volume[current_FCB_pos + 23] = mod_time;
      
      mod_time++;

    }
    // printf("start block is %d  \n ", start_block);
    // printf("in open is .. %c \n", fs->volume[4096]);
    return start_block;
  }
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	
  for (int i = 0; i < size; ++i){
    output[i] = fs->volume[fp * 32 + i + fs->FILE_BASE_ADDRESS];
  }

}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
  
  /* if enough space to write */
  if ( IsEnoughSpace(fs,fp,size) ){

    for (int i = 0; i < size; ++i){
      /* update the disk */
      fs->volume[fs->FILE_BASE_ADDRESS + fp * 32 + i] = input[i];
      /* update the super block */
      if ( i % 32 == 0){
        u32 super_block = fp + i/32;
        int shiftnum = super_block % 8;
        fs->volume[super_block/8] = fs->volume[super_block/8] + (1 << shiftnum); // modify one bit
      }
    }

    u32 pre_file_size =  (fs->volume[current_FCB_pos + 28] << 24) + (fs->volume[current_FCB_pos + 29] << 16)
                        +(fs->volume[current_FCB_pos + 30] << 8) + (fs->volume[current_FCB_pos + 31]);
    
    /**/
    u32 delta_size = pre_file_size - size;
    
    
    if ((int) delta_size < 0 ){
      block_pos = block_pos + (-delta_size - 1)/32 + 1;
    }

    /* update the size */
    fs->volume[current_FCB_pos + 28] = size >> 24;
    fs->volume[current_FCB_pos + 29] = size >> 16;
    fs->volume[current_FCB_pos + 30] = size >> 8;
    fs->volume[current_FCB_pos + 31] = size;

    


    if (delta_size > 0 && pre_file_size != 0 && fp != block_pos - 1){
      Update(fs, fp + (size -1)/32 + 1 , delta_size);
    }
    // printf(" current block is %d \n", block_pos);
    
  }

  /* not enough space */
  else{
    
    if (block_pos * 32 - 1 + size >= fs->SUPERBLOCK_SIZE){
      printf("Write Error: you Write the file out of limited space \n");
      return -1;
    }

    /* write the file into new space */
    for ( int i = 0; i < size; ++i){

      fs->volume[fs->FILE_BASE_ADDRESS + block_pos * 32 + i] = input[i];

      /* update the super block */
      if ( i % 32 == 0){
        u32 super_block = block_pos + i/32;
        int shiftnum = super_block % 8;
        fs->volume[super_block/8] = fs->volume[super_block/8] + (1 << shiftnum); // modify one bit
      }
    }

    /* update the size */
    fs->volume[current_FCB_pos + 28] = size >> 24;
    fs->volume[current_FCB_pos + 29] = size >> 16;
    fs->volume[current_FCB_pos + 30] = size >> 8;
    fs->volume[current_FCB_pos + 31] = size;

    /* uodate the start position */
    fs->volume[current_FCB_pos + 24] = block_pos >> 24;
    fs->volume[current_FCB_pos + 25] = block_pos >> 16;
    fs->volume[current_FCB_pos + 26] = block_pos >> 8;
    fs->volume[current_FCB_pos + 27] = block_pos;

    u32 pre_file_size = (fs->volume[current_FCB_pos + 28] << 24) + (fs->volume[current_FCB_pos + 29] << 16) 
                        +(fs->volume[current_FCB_pos + 30] << 8) + (fs->volume[current_FCB_pos + 31]);
    
    Update(fs, fp , pre_file_size);
  }

}

__device__ void swap(FileSystem * fs, u32 pre, u32 after){
  for (int i = 0; i < 32; i++){
    uchar temp = fs->volume[pre + i];
		fs->volume[pre + i] = fs->volume[after + i];
		fs->volume[after + i] = temp;
  }
}

__device__ void Sort(FileSystem *  fs, u32 begin, u32 end, int op){
  
  
  if (op == 0){
    for ( int i = begin; i < end + 32; i += 32){
      for ( int j = begin; j < end - i + begin + 32; j += 32 ){
        u32 previous_date =  (fs->volume[j + 22] << 8) + (fs->volume[j + 23]);
				u32 after_date = (fs->volume[j + 22 + 32] << 8) + (fs->volume[j + 23 + 32]);
				if ( previous_date < after_date ) swap(fs, j, j + 32);
        
      }
    }
  }

  else{
    for (int i = begin; i < end + 32; i += 32 ){
      for ( int j = begin; j < end -i + begin + 32; j += 32){
        u32 pre_size = (fs->volume[j + 28] << 24) + (fs->volume[j + 29] << 16)
                      +(fs->volume[j + 30] << 8)  + (fs->volume[j + 31]);
        u32 after_size =  (fs->volume[j + 28 + 32] << 24) + (fs->volume[j + 29 + 32] << 16)
                          +(fs->volume[j + 30 + 32] << 8)  + (fs->volume[j + 31 + 32]);

        u32 pre_creadate =  (fs->volume[j + 20] << 8) + (fs->volume[j + 21]);
        u32 after_creadate = (fs->volume[j + 20 + 32] << 8) + (fs->volume[j + 21 + 32]);

        if (pre_size < after_size || (pre_size == after_size && pre_creadate > after_creadate )){
          swap(fs, j , j+32);
        }
        
      }
    }
  }
}

__device__ void Print(FileSystem * fs, u32 stop_pos, int op){
  char result[20];

  if ( op == 0){
    printf("===sort by modified time===\n");
    for (int i = 4096; i < stop_pos + 32; i+=32){
      for (int j = 0; j < 20; j++){
        result[j] = fs->volume[j+i];
      }
      printf("%s\n",result);
    }
  }


  else{
    u32 temp;
    printf("===sort by file size===\n");
    for (int i = 4096; i < stop_pos + 32; i+=32){
      for (int j = 0; j < 20; j++){
        result[j] = fs->volume[j+i];
      }
      temp = (fs->volume[i+28] << 24) + (fs->volume[i+29] << 16)
            +(fs->volume[i+30] << 8) + (fs->volume[i+31]);

      printf("%s %d \n", result,temp );
    }
  }

}


__device__ void fs_gsys(FileSystem *fs, int op)
{
	u32 stop_pos;

  /* search the stop point */
  for ( int i = 4096 ; i < (4096 + 32*1024 - 1); i += 32 ){
    u32 file_size = (fs->volume[i + 28] <<  24) + (fs->volume[i + 29] <<  16) 
                   +(fs->volume[i + 30] <<  8)  + (fs->volume[i + 31]);

    if (file_size == 0) break;
    stop_pos = i ;
  }


  if (stop_pos <  4096){
    printf("LS Error: No file in FCB \n");
  }

  Sort(fs, 4096, stop_pos, op);
  Print(fs, stop_pos, op);


}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
  u32 ISExist = IsFileExist(fs, s);

  if ((int) ISExist == -1){
    printf("Remove Error: No Such file! \n");
    return;
  }

  else{
    /* search*/
    current_FCB_pos = ISExist;

    u32 start_block = (fs->volume[current_FCB_pos + 24] << 24) + (fs->volume[current_FCB_pos + 25] << 16)
                    + (fs->volume[current_FCB_pos + 26] << 8) + (fs->volume[current_FCB_pos + 27]);

    u32 file_size = (fs->volume[current_FCB_pos + 28] << 24) + (fs->volume[current_FCB_pos + 29] << 16)
                  + (fs->volume[current_FCB_pos + 30] << 8) + (fs->volume[current_FCB_pos + 31]);

    u32 block_size = (file_size - 1) / 32 + 1;
    for (int i = 0; i < block_size; i++){
      fs->volume[start_block + i ] = 0;
    }

    for (int i = 0; i < file_size; i ++){
      fs->volume[start_block * 32 + i + fs->FILE_BASE_ADDRESS] = 0;
    }

    Update(fs, start_block, file_size);

    for (int i = current_FCB_pos; i < (4096 + 32*1024 - 1); i += 32){
      u32 size = (fs->volume[i + 28] << 24) + (fs->volume[i + 29] << 16)
                +(fs->volume[i + 30] << 8) + (fs->volume[i + 31]);
      if (size == 0) break;
      for (int j = 0; j < 32; j++){
        fs->volume[i + j] = fs->volume[i + j + 32];
        fs->volume[i + j + 32] = 0;
      }
    }

    FCB_pos -= 32;
  }

  
}
